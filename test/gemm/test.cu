#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <gemm_core/gemm_core.hpp>

constexpr unsigned N = 16;
constexpr unsigned K = 32;

template <class T>
std::string get_type_name();
template <> std::string get_type_name<double>(){return "double";}
template <> std::string get_type_name<float>(){return "float";}
template <> std::string get_type_name<half>(){return "half";}

template <class T, class S>
__device__ __host__ T convert(const S a) {return static_cast<T>(a);}
template <> __device__ __host__ float convert<float, float>(const float a) {return a;}
template <> __device__ __host__ float convert<float, half >(const half  a) {return __half2float(a);}
template <> __device__ __host__ half  convert<half , float>(const float a) {return __float2half(a);}
template <> __device__ __host__ half  convert<half , half >(const half  a) {return a;}

template <class T, unsigned K>
__global__ void test_gemm_16x16_kernel(T* const c, const T* const a, const T* const b){
	mtk::gemm_core::gemm_core16x16<K>(c, N, a, N, b, K, threadIdx.x & 0x1f);
}

template <class T>
void test_gemm(){
	T* a;
	T* b;
	T* c;
	T* d;

	std::printf("%s\n", get_type_name<T>().c_str());

	hipHostMalloc(&a, N * K * sizeof(T));
	hipHostMalloc(&b, K * N * sizeof(T));
	hipHostMalloc(&c, N * N * sizeof(T));
	hipHostMalloc(&d, N * N * sizeof(T));

	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

	for(unsigned i = 0; i < K * N; i++){
		a[i] = convert<T>(dist(mt));
	}
	for(unsigned i = 0; i < K * N; i++){
		b[i] = convert<T>(dist(mt));
	}
	for(unsigned i = 0; i < N * N; i++){
		d[i] = c[i] = convert<T>(dist(mt));
	}

	hipDeviceSynchronize();
	test_gemm_16x16_kernel<T, K><<<1, 32>>>(d, a, b);
	hipDeviceSynchronize();

	double error = 0.0;
	for(unsigned i = 0; i < N; i++){
		for(unsigned j = 0; j < N; j++){
			double sum = c[i + j * N];
			for(unsigned k = 0; k < K; k++){
				sum += convert<double>(a[k * N + i]) * convert<double>(b[j * K + k]);
			}
			error = std::max(error, std::abs(convert<double>(d[i + j * N]) - sum));
		}
	}
	std::printf("error = %e\n", error);

	hipFree(a);
	hipFree(b);
	hipFree(c);
	hipFree(d);
}

int main() {
	test_gemm<double>();
	test_gemm<float>();
	test_gemm<half >();
}
