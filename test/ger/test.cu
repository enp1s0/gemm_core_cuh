#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <gemm_core.cuh>

constexpr unsigned N = 16;

template <class T>
std::string get_type_name();
template <> std::string get_type_name<float>(){return "float";}
template <> std::string get_type_name<half>(){return "half";}

template <class T, class S>
__device__ __host__ T convert(const S);
template <> __device__ __host__ float convert<float, float>(const float a) {return a;}
template <> __device__ __host__ float convert<float, half >(const half  a) {return __half2float(a);}
template <> __device__ __host__ half  convert<half , float>(const float a) {return __float2half(a);}
template <> __device__ __host__ half  convert<half , half >(const half  a) {return a;}

template <class T>
__global__ void test_gemv_16x16_kernel(T* const c, const T* const a, const T* const b){
	mtk::ger_core16x16<T, 1>(c, N, a, b, threadIdx.x & 0x1f);
}

template <class T>
void test_gemv(){
	T* a;
	T* b;
	T* c;

	std::printf("%s\n", get_type_name<T>().c_str());

	hipHostMalloc(&a, N * sizeof(T));
	hipHostMalloc(&b, N * sizeof(T));
	hipHostMalloc(&c, N * N * sizeof(T));

	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

	for(unsigned i = 0; i < N; i++){
		a[i] = convert<T>(dist(mt));
	}
	for(unsigned i = 0; i < N; i++){
		b[i] = convert<T>(dist(mt));
	}

	hipDeviceSynchronize();
	test_gemv_16x16_kernel<T><<<1, 32>>>(c, a, b);
	hipDeviceSynchronize();

	float error = 0.0f;
	for(unsigned i = 0; i < N; i++){
		for(unsigned j = 0; j < N; j++){
			const auto ca = convert<float>(a[i]) * convert<float>(b[j]);
			error = std::max((convert<float>(c[i + j * N]) - ca) * (convert<float>(c[i + j * N]) - ca), error);
		}
	}
	std::printf("error = %e\n", std::sqrt(error));

	hipFree(a);
	hipFree(b);
	hipFree(c);
}

int main() {
	test_gemv<float>();
	test_gemv<half >();
}
