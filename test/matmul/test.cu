#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <gemm_core.cuh>

constexpr unsigned N = 16;

template <class T>
std::string get_type_name();
template <> std::string get_type_name<float>(){return "float";}
template <> std::string get_type_name<half>(){return "half";}

template <class T, class S>
__device__ __host__ T convert(const S);
template <> __device__ __host__ float convert<float, float>(const float a) {return a;}
template <> __device__ __host__ float convert<float, half >(const half  a) {return __half2float(a);}
template <> __device__ __host__ half  convert<half , float>(const float a) {return __float2half(a);}
template <> __device__ __host__ half  convert<half , half >(const half  a) {return a;}

template <class T>
__global__ void test_gemv_16x16_kernel(T* const c, const T* const a, const T* const b){
	mtk::matmul_core16x16(c, N, a, N, b, N, threadIdx.x & 0x1f);
}

template <class T>
void test_gemv(){
	T* a;
	T* b;
	T* c;

	std::printf("%s\n", get_type_name<T>().c_str());

	hipHostMalloc(&a, N * N * sizeof(T));
	hipHostMalloc(&b, N * N * sizeof(T));
	hipHostMalloc(&c, N * N * sizeof(T));

	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

	for(unsigned i = 0; i < N * N; i++){
		a[i] = convert<T>(dist(mt));
	}
	for(unsigned i = 0; i < N * N; i++){
		b[i] = convert<T>(dist(mt));
	}
	for(unsigned i = 0; i < N * N; i++){
		c[i] = convert<T>(0.0f);
	}

	hipDeviceSynchronize();
	test_gemv_16x16_kernel<T><<<1, 32>>>(c, a, b);
	hipDeviceSynchronize();

	float error = 0.0f;
	for(unsigned i = 0; i < N; i++){
		for(unsigned j = 0; j < N; j++){
			float sum = 0.0f;
			for(unsigned k = 0; k < N; k++){
				sum += convert<float>(a[k * N + i]) * convert<float>(b[j * N + k]);
			}
			error = std::max(error, std::abs(convert<float>(c[i + j * N]) - sum));
		}
	}
	std::printf("error = %e\n", error);

	hipFree(a);
	hipFree(b);
	hipFree(c);
}

int main() {
	test_gemv<float>();
	test_gemv<half >();
}
