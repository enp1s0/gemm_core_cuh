#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <chrono>
#include <cutf/memory.hpp>
#include <cutf/error.hpp>
#include "gemm_core.cuh"

constexpr std::size_t m = (1 << 13) - 1;
constexpr std::size_t n = (1 << 13) - 1;
constexpr std::size_t k = (1 << 13) - 1;

constexpr std::size_t warp_size = 32;
constexpr std::size_t block_size = 256;

void print_gemm_info(const std::size_t m, const std::size_t n, const std::size_t k, const std::size_t grid_size, std::size_t block_size, double elapsed_time){
	std::cout<<"Matrix size : "<<m<<", "<<n<<", "<<k<<std::endl;
	std::cout<<"Grid size   : "<<grid_size<<std::endl;
	std::cout<<"Block size  : "<<block_size<<std::endl;
	std::cout<<"Performance : "<<(m * n * k * 2 / elapsed_time / (1024 * 1024 * 1024)) <<" GFLOPS"<<std::endl;
}

template <class T, unsigned num_warps>
__global__ void test_gemm_16x16_kernel(T* const c, const T* const a, const T* const b, const std::size_t m, const std::size_t n, const std::size_t k){}

template <>
__global__ void test_gemm_16x16_kernel<float, 1>(float* const c, const float* const a, const float* const b, const std::size_t m, const std::size_t n, const std::size_t k){
	constexpr std::size_t num_blocks_per_grid = block_size / warp_size;
	const std::size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	const auto num_m_blocks = (m + 15) / 16;
	const auto num_n_blocks = (n + 15) / 16;
	const auto num_k_blocks = (k + 15) / 16;
	const auto matrix_id = tid / warpSize;

	const auto block_m = matrix_id / (num_n_blocks * num_k_blocks);
	const auto block_n = (matrix_id % (num_n_blocks * num_k_blocks)) / num_k_blocks;
	const auto block_k = (matrix_id % (num_n_blocks * num_k_blocks)) % num_k_blocks;

	__shared__ float shared_a[16 * 16 * num_blocks_per_grid];
	__shared__ float shared_b[16 * 16 * num_blocks_per_grid];
	__shared__ float shared_c[16 * 16 * num_blocks_per_grid];

	float *const shared_a_ptr = shared_a + 16 * 16 * (tid >> 5);
	float *const shared_b_ptr = shared_b + 16 * 16 * (tid >> 5);
	float *const shared_c_ptr = shared_c + 16 * 16 * (tid >> 5);

	// Load
	
	gemm_core16x16<float, 1>(shared_c_ptr, shared_a_ptr, shared_b_ptr, tid & 0xf1);

	// Store
}

template <class Func>
double get_elapsed_time(Func func){
	const auto start = std::chrono::system_clock::now();
	func();
	const auto end = std::chrono::system_clock::now();
	return std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000000.0;
}


template <class T, unsigned num_warps>
void test_gemm_16x16(T* const c, const T* const a, const T* const b, const std::size_t m, const std::size_t n, const std::size_t k){}

template <>
void test_gemm_16x16<float, 1>(float* const c, const float* const a, const float* const b, const std::size_t m, const std::size_t n, const std::size_t k){
	const auto num_m_blocks = (m + 15) / 16;
	const auto num_n_blocks = (n + 15) / 16;
	const auto num_k_blocks = (k + 15) / 16;

	const auto num_threads = (num_m_blocks * num_n_blocks * num_k_blocks) * warp_size;
	const auto grid_size = num_threads / block_size;

	const auto elapsed_time = get_elapsed_time(
			[&a, &b, &c, &m, &n, &k, &grid_size](){
			test_gemm_16x16_kernel<float, 1><<<grid_size, block_size>>>(c, a, b, m, n, k);
			hipDeviceSynchronize();
			});

	print_gemm_info(m, n, k, grid_size, block_size, elapsed_time);
}

int main(){
	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
	auto d_a = cutf::memory::get_device_unique_ptr<float>(m * k);
	auto d_b = cutf::memory::get_device_unique_ptr<float>(k * n);
	auto d_c = cutf::memory::get_device_unique_ptr<float>(m * n);
	auto h_a = cutf::memory::get_host_unique_ptr<float>(m * k);
	auto h_b = cutf::memory::get_host_unique_ptr<float>(k * n);
	auto h_c = cutf::memory::get_host_unique_ptr<float>(m * n);

#pragma omp parallel for
	for(std::size_t i = 0; i < m * k; i++) h_a.get()[i] = dist(mt);
#pragma omp parallel for
	for(std::size_t i = 0; i < k * n; i++) h_b.get()[i] = dist(mt);
#pragma omp parallel for
	for(std::size_t i = 0; i < m * n; i++) h_c.get()[i] = 0.0f;

	cutf::memory::copy(d_a.get(), h_a.get(), m * k);
	cutf::memory::copy(d_b.get(), h_b.get(), k * n);
	cutf::memory::copy(d_c.get(), h_c.get(), m * n);

	test_gemm_16x16<float, 1>(d_c.get(), d_a.get(), d_b.get(), m, n, k);

	cutf::memory::copy(d_c.get(), h_c.get(), m * n);
}
