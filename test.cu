#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <chrono>
#include <cutf/cublas.hpp>
#include <cutf/memory.hpp>
#include <cutf/error.hpp>
#include <cutf/type.hpp>
#include "gemm_core.cuh"

constexpr std::size_t m = (1 << 15) - 1;
constexpr std::size_t n = (1 << 15) - 1;
constexpr std::size_t k = (1 << 11) - 1;

constexpr std::size_t warp_size = 32;
constexpr std::size_t block_size = 512;

template <class T>
__device__ __host__ inline void print_matrix(const T* const ptr, std::size_t m, std::size_t n, const char *name = nullptr){
	if(name != nullptr) printf("%s = \n", name);
	for(int i = 0; i < m; i++){
		for(int j = 0; j < n; j++){
			const auto val = cutf::type::cast<float>(ptr[j * m + i]);
			if(val < 0.0f){
				printf("%.5f ", val);
			}else{
				printf(" %.5f ", val);
			}
		}
		printf("\n");
	}
}


template <class T>
void print_gemm_info(const std::size_t m, const std::size_t n, const std::size_t k, const std::size_t grid_size, std::size_t block_size, double elapsed_time){
	std::cout<<"Matrix size : "<<m<<", "<<n<<", "<<k<<std::endl;
	std::cout<<"Memory      : "<<((m * n + n * k + k * m) * sizeof(T) / (1024.0 * 1024.0))<<" MB"<<std::endl;
	std::cout<<"Grid size   : "<<grid_size<<std::endl;
	std::cout<<"Block size  : "<<block_size<<std::endl;
	std::cout<<"Elapsed time: "<<elapsed_time<<" [s]"<<std::endl;
	std::cout<<"Performance : "<<(m * n * k * 2 / elapsed_time / (1024.0 * 1024.0 * 1024.0 * 1024.0)) <<" TFLOPS"<<std::endl;
}

template <class T, unsigned num_warps>
__device__ void load64x64(
		T* const dst,
		const T* const src, const std::size_t m, const std::size_t n,
		const std::size_t start_m, const std::size_t start_n,
		const unsigned unique_id, const unsigned warp_id
		){
	constexpr std::size_t dim = 64;
	if(start_m + dim >= m || start_n + dim >= n){
		for(unsigned i = warp_id; i < dim; i+=num_warps){
			const auto load_n = start_n + i;

			for(unsigned j = 0; j < dim; j += warp_size){
				const auto load_m = start_m + j + unique_id;
				T tmp = cutf::type::cast<T>(0.0f);
				if(load_m < m && load_n < n){
					tmp = __ldg( &src[load_m + load_n * m] );
				}

				dst[j + unique_id + i * dim] = tmp;
			}
		}
	}else{
#pragma unroll
		for(unsigned i = warp_id; i < dim; i+=num_warps){
			const auto load_n = start_n + i;

#pragma unroll
			for(unsigned j = 0; j < dim; j += warp_size){
				const auto load_m = start_m + j + unique_id;
				
				dst[j + unique_id + i * dim] = __ldg( &src[load_m + load_n * m]);
			}
		}
	}
}

template <class T, unsigned num_warps>
__device__ void store64x64(
		T* const dst,const std::size_t m, const std::size_t n,
		const std::size_t start_m, const std::size_t start_n,
		const T* const src, 
		const unsigned unique_id, const unsigned warp_id
		){
	constexpr std::size_t dim = 64;
	if(start_m + dim >= m || start_n + dim >= n){
		for(unsigned i = warp_id; i < dim; i+=num_warps){
			const auto load_n = start_n + i;

			for(unsigned j = 0; j < dim; j += warp_size){
				const auto load_m = start_m + j + unique_id;
				if(load_m < m && load_n < n){
					dst[load_m + load_n * m] = src[j + unique_id + i * dim];
				}

			}
		}
	}else{
#pragma unroll
		for(unsigned i = warp_id; i < dim; i+=num_warps){
			const auto load_n = start_n + i;

#pragma unroll
			for(unsigned j = 0; j < dim; j += warp_size){
				const auto load_m = start_m + j + unique_id;
				
				dst[load_m + load_n * m] = src[j + unique_id + i * dim];
			}
		}
	}
}

template <class T>
__global__ void test_gemm_16x16_kernel(T* const c, const T* const a, const T* const b, const std::size_t m, const std::size_t n, const std::size_t k){}

template <>
__global__ void test_gemm_16x16_kernel<float>(float* const c, const float* const a, const float* const b, const std::size_t m, const std::size_t n, const std::size_t k){
	constexpr std::size_t dim = 64;
	const auto num_m_blocks = (m + dim - 1) / dim;
	const auto num_k_blocks = (k + dim - 1) / dim;
	const auto matrix_id = blockIdx.x;
	const unsigned unique_id = threadIdx.x & (warp_size - 1); 
	const unsigned warp_id = threadIdx.x >> 5;

	const std::size_t block_m = matrix_id % num_m_blocks;
	const std::size_t block_n = matrix_id / num_m_blocks;

	__shared__ float shared_a[16 * 16 * 4 * 4];
	__shared__ float shared_b[16 * 16 * 4 * 4];
	__shared__ float shared_c[16 * 16 * 4 * 4];


	for(std::size_t ik = 0; ik < num_k_blocks; ik++){
		// Load C
		const auto block_m_start = block_m * dim;
		const auto block_n_start = block_n * dim;
		const auto block_k_start = ik * dim;
		load64x64<float, (block_size/warp_size)>(shared_c,
				c, m, n,
				block_m_start, block_n_start,
				unique_id, warp_id);
		// Load A
		load64x64<float, (block_size/warp_size)>(shared_a,
				a, m, k,
				block_m_start, block_k_start,
				unique_id, warp_id);
		// Load B
		load64x64<float, (block_size/warp_size)>(shared_b,
				b, k, n,
				block_k_start, block_n_start,
				unique_id, warp_id);

		__syncthreads();

		constexpr unsigned num_blocks_per_grid = block_size / warp_size;
		for(unsigned i = 0; i < 16 / num_blocks_per_grid; i++){
			const auto sub_block_m = 2 * i + (warp_id / 4);
			const auto sub_block_n = warp_id & (dim/16 - 1);
			for(unsigned j = 0; j < (dim/16); j++){
				gemm_core16x16<float, 1>(
						shared_c + sub_block_n * dim * 16 + sub_block_m * 16,
						shared_a + sub_block_m * 16 + j * (dim * 16),
						shared_b + j * 16 + sub_block_n * (dim * 16),
						dim, unique_id);
			}
		}

		__syncthreads();

		// Store C
		store64x64<float, (block_size/warp_size)>(
				c, m, n,
				block_m * dim, block_n * dim,
				shared_c,
				unique_id, warp_id
				);
	}
}

template <class Func>
double get_elapsed_time(Func func){
	const auto start = std::chrono::system_clock::now();
	func();
	const auto end = std::chrono::system_clock::now();
	return std::chrono::duration_cast<std::chrono::microseconds>(end - start).count() / 1000000.0;
}


template <class T, unsigned num_warps>
void test_gemm_16x16(T* const c, const T* const a, const T* const b, const std::size_t m, const std::size_t n, const std::size_t k){}

template <>
void test_gemm_16x16<float, 1>(float* const c, const float* const a, const float* const b, const std::size_t m, const std::size_t n, const std::size_t k){
	constexpr std::size_t dim = 64;
	constexpr std::size_t C = 1;
	const auto num_m_blocks = (m + dim - 1) / dim;
	const auto num_n_blocks = (n + dim - 1) / dim;

	const auto grid_size = num_n_blocks * num_m_blocks;

	const auto elapsed_time = get_elapsed_time(
			[&a, &b, &c, &m, &n, &k, &grid_size](){
			for(std::size_t i = 0;i < C; i++)
			test_gemm_16x16_kernel<float><<<grid_size, block_size>>>(c, a, b, m, n, k);
			CUTF_HANDLE_ERROR(hipDeviceSynchronize());
			});

	print_gemm_info<float>(m, n, k, grid_size, block_size, elapsed_time / C);
}

int main(){
	std::mt19937 mt(std::random_device{}());
	std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
	auto d_a = cutf::memory::get_device_unique_ptr<float>(m * k);
	auto d_b = cutf::memory::get_device_unique_ptr<float>(k * n);
	auto d_c = cutf::memory::get_device_unique_ptr<float>(m * n);
	auto h_a = cutf::memory::get_host_unique_ptr<float>(m * k);
	auto h_b = cutf::memory::get_host_unique_ptr<float>(k * n);
	auto h_c = cutf::memory::get_host_unique_ptr<float>(m * n);

#pragma omp parallel for
	for(std::size_t i = 0; i < m * k; i++) h_a.get()[i] = dist(mt);
#pragma omp parallel for
	for(std::size_t i = 0; i < k * n; i++) h_b.get()[i] = dist(mt);
#pragma omp parallel for
	for(std::size_t i = 0; i < m * n; i++) h_c.get()[i] = 0.0f;

	cutf::memory::copy(d_a.get(), h_a.get(), m * k);
	cutf::memory::copy(d_b.get(), h_b.get(), k * n);
	cutf::memory::copy(d_c.get(), h_c.get(), m * n);

	test_gemm_16x16<float, 1>(d_c.get(), d_a.get(), d_b.get(), m, n, k);

	cutf::memory::copy(h_c.get(), d_c.get(), m * n);
	float c_norm = 0.0f;
	for(std::size_t i = 0; i < m * n; i++){
		c_norm += h_c.get()[i] * h_c.get()[i];
	}

	// Validation
	auto cublas = cutf::cublas::get_cublas_unique_ptr();
	float alpha = 1.0f, beta = -1.0f;
	CUTF_HANDLE_ERROR(
			cutf::cublas::gemm(*cublas.get(),
				HIPBLAS_OP_N, HIPBLAS_OP_N,
				m, n, k,
				&alpha,
				d_a.get(), m,
				d_b.get(), k,
				&beta,
				d_c.get(), m
			));
	cutf::memory::copy(h_c.get(), d_c.get(), m * n);

	float error = 0.0f;
	for(std::size_t i = 0; i < m * n; i++){
		error += h_c.get()[i] * h_c.get()[i];
	}

	std::cout<<"Error    : "<<std::sqrt(error/c_norm)<<std::endl;
}
